#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void upSweep(int n, int* data, int offset)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;
            if (idx % offset == 0)
            {
                data[idx + offset - 1] += data[idx + (offset >> 1) - 1];
            }
        }

        __global__ void downSweep(int n, int* data, int offset)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;
            if (idx % offset == 0)
            {
                int temp = data[idx + (offset >> 1) - 1];
                data[idx + (offset >> 1) - 1] = data[idx + offset - 1];
                data[idx + offset - 1] += temp;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {
            // Calculate necessary padding for non-powers of 2, if power of two then == 0
            int nPadded = 1 << ilog2ceil(n);

            // Declare and allocate padded buffer
            int* dev_in;
            hipMalloc((void**)&dev_in, nPadded * sizeof(int));
            checkCUDAError("Cuda Malloc");

            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Cuda memcpy host to device");

            // Set padded memory to zeroes
            hipMemset(dev_in + n, 0, (nPadded - n) * sizeof(int));

            int blockSize = 128;
            int blocks = (nPadded + blockSize - 1) / blockSize;

            timer().startGpuTimer();
            // Up-sweep phase
            for (int i = 0; i < ilog2ceil(n); i++)
            {
                int offset = 1 << (i + 1);
                upSweep << <blocks, blockSize >> > (nPadded, dev_in, offset);
            }

            // Set last element to zero
            hipMemset(dev_in + (nPadded - 1), 0, sizeof(int));
            checkCUDAError("Cuda memset between up and down sweep");

            // Down-sweep phase
            for (int i = ilog2ceil(n) - 1; i >= 0; i--)
            {
                int offset = 1 << (i + 1);
                downSweep << <blocks, blockSize >> > (nPadded, dev_in, offset);
            }

            hipMemcpy(odata, dev_in, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Cuda memcpy device to host");

            timer().endGpuTimer();
        }

        __global__ void upSweepOpt(int n, int* data, int offset)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            int numNodes = n / offset;
        	if (idx >= numNodes) return;

        	int right = (idx + 1) * offset - 1;
            int left = right - (offset >> 1);

            if (left < 0) return;
            data[right] += data[left];
        }

        __global__ void downSweepOpt(int n, int* data, int offset)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            int numNodes = n / offset;
        	if (idx >= numNodes) return;

        	int right = (idx + 1) * offset - 1;
            int left = right - (offset >> 1);

            if (left < 0) return;
            int temp = data[left];
            data[left] = data[right];
            data[right] += temp;
        
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scanOpt(int n, int *odata, const int *idata) {
            // Calculate necessary padding for non-powers of 2, if power of two then == 0
            int nPadded = 1 << ilog2ceil(n);

            // Declare and allocate padded buffer
            int* dev_in;
            hipMalloc((void**)&dev_in, nPadded * sizeof(int));
            checkCUDAError("Cuda Malloc");

            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Cuda memcpy host to device");

            // Set padded memory to zeroes
            hipMemset(dev_in + n, 0, (nPadded - n) * sizeof(int));
            checkCUDAError("Cuda memset dev_in padding");

            int blockSize = 256;

            timer().startGpuTimer();
            // Up-sweep phase
            for (int i = 0; i < ilog2ceil(n); i++)
            {
                int offset = 1 << (i + 1);
                int numNodes = nPadded / offset;
                int actualBlocks = (numNodes + blockSize - 1) / blockSize;
                upSweepOpt << <actualBlocks, blockSize >> > (nPadded, dev_in, offset);
                checkCUDAError("upSweep launch");
            }

            // Set last element to zero
            hipMemset(dev_in + (nPadded - 1), 0, sizeof(int));
            checkCUDAError("Cuda memset between up and down sweep");

            // Down-sweep phase
            for (int i = ilog2ceil(n) - 1; i >= 0; i--)
            {
                int offset = 1 << (i + 1);
                int numNodes = nPadded / offset;
                int actualBlocks = (numNodes + blockSize - 1) / blockSize;
                downSweepOpt << <actualBlocks, blockSize >> > (nPadded, dev_in, offset);
                checkCUDAError("downSweep launch");
            }
            hipDeviceSynchronize();
            timer().endGpuTimer();

            hipMemcpy(odata, dev_in, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Cuda memcpy device to host");

            hipFree(dev_in);
            checkCUDAError("Cuda free dev_in");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int nPadded = 1 << ilog2ceil(n);
            // Allocate global memory
            int* dev_in;
            int* dev_bools;
            int* dev_scanned;
            int* dev_out;

            hipMalloc((void**)&dev_in, n * sizeof(int));
            checkCUDAError("Cuda malloc dev_in");
            hipMalloc((void**)&dev_bools, nPadded * sizeof(int));
            checkCUDAError("Cuda malloc dev_bools");
            hipMalloc((void**)&dev_scanned, nPadded * sizeof(int));
            checkCUDAError("Cuda malloc dev_scanned");
            hipMalloc((void**)&dev_out, n * sizeof(int));
            checkCUDAError("Cuda malloc dev_out");

            int blockSize = 256;
            int blocks = (n + blockSize - 1) / blockSize;

            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("Cuda memcpy dev_in  host to device");

            // Pre-zero padding for exclusive scan result
            hipMemset(dev_scanned + n, 0, (nPadded - n) * sizeof(int));
            checkCUDAError("Cuda memset dev_scanned padding");

        	timer().startGpuTimer();

            // Map input to booleans
            StreamCompaction::Common::kernMapToBoolean << <blocks, blockSize >> > (n, dev_bools, dev_in);

            // Copy bool values to dev_scanned so we can do in-place scan without losing bools
            hipMemcpy(dev_scanned, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("Cuda memcpy dev_bools to dev_scanned");

            // Up-sweep phase
            for (int i = 0; i < ilog2ceil(n); i++)
            {
                int offset = 1 << (i + 1);
                int numNodes = nPadded / offset;
                int actualBlocks = (numNodes + blockSize - 1) / blockSize;
                upSweepOpt << <actualBlocks, blockSize >> > (nPadded, dev_scanned, offset);
                checkCUDAError("upSweep launch");
            }

            // Set last element to zero
            hipMemset(dev_scanned + (nPadded - 1), 0, sizeof(int));
            checkCUDAError("Cuda memset between up and down sweep");

            // Down-sweep phase
            for (int i = ilog2ceil(n) - 1; i >= 0; i--)
            {
                int offset = 1 << (i + 1);
                int numNodes = nPadded / offset;
                int actualBlocks = (numNodes + blockSize - 1) / blockSize;
                downSweepOpt << <actualBlocks, blockSize >> > (nPadded, dev_scanned, offset);
                checkCUDAError("downSweep launch");
            }

            // Before scatter, get scan result + last boolean for return
            int lastTwo[2];
            hipMemcpy(lastTwo, dev_scanned + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Cuda memcpy exclusive scan result");
        	hipMemcpy(lastTwo + 1, dev_bools + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
        	checkCUDAError("Cuda memcpy last boolean");

            int scatterRemaining = lastTwo[0] + lastTwo[1];

            // Scatter step
            StreamCompaction::Common::kernScatter<<<blocks, blockSize>>>(n, dev_out, dev_in, dev_bools, dev_scanned);
            hipDeviceSynchronize();
        	timer().endGpuTimer();

            // Bring results to host
            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("Cuda memcpy to odata device to host");

            // Free memory
            hipFree(dev_in);
            checkCUDAError("Cuda free dev_in");
            hipFree(dev_bools);
            checkCUDAError("Cuda free dev_bools");
            hipFree(dev_scanned);
            checkCUDAError("Cuda free dev_scanned");
            hipFree(dev_out);
            checkCUDAError("Cuda free dev_out");

            return scatterRemaining;
        }
    }
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        
        __global__ void scanStep(int n, int* odata, const int* idata, int offset) {
            int idx = threadIdx.x + blockIdx.x * blockDim.x;
            if (idx >= n) return;
            if (idx >= offset)
            {
                odata[idx] = idata[idx - offset] + idata[idx];
            } else
            {
                odata[idx] = idata[idx];
            }
            
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // Declare and allocate global memory
            int *dev_in, *dev_out;
            hipMalloc((void**)&dev_in, n * sizeof(int));
            hipMalloc((void**)&dev_out, n * sizeof(int));

            // Copy data to device
            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 128;
            int blocks = (n + blockSize - 1) / blockSize;

            timer().startGpuTimer();
            for (int i = 0; i < ilog2ceil(n); i++)
            {
                int offset = 1 << (i - 1);
	            scanStep<<<blocks, blockSize>>>(n)
            }
            timer().endGpuTimer();

            // Copy result back to CPU
            hipMemcpy(odata, dev_in, n * sizeof(int), hipMemcpyDeviceToHost);
            // Free global memory
            hipFree(dev_in);
            hipFree(dev_out);
        }
    }
}
